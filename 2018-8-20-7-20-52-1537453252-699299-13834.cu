//This is a generated CUDA code

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>

//f_alu = 1
//f_mad =8
//f_sfu =6
//d_alu =8
//d_mad =6
//b_alu =5
__constant__ float kconst[115] = {
2640.27049419,496.788317279,755.85277182,4070.19741521,6510.34703498,2039.14289025,3704.61925152,7755.66914948,
1861.26002473,1253.44798987,2629.46951817,7940.16158665,7767.42297282,5605.65608719,5373.03014677,3891.71871656,
366.965906878,8379.0600811,3930.37788456,8751.34651216,6043.60764794,9682.30745623,6178.41261724,4564.3931025,
374.255925925,4577.84509961,1671.08086172,9345.83987891,2901.2040663,4154.98833481,8208.38540292,4352.1328191,
9573.36924331,3580.73886635,269.083439352,352.007680208,5305.30563288,3596.20743502,438.919179395,9456.31492571,
8206.88128285,7265.8087816,8790.45564504,2464.18248902,4064.78800258,6832.31591731,9795.16429237,5786.85994295,
2010.51460232,4034.59881526,6585.45210739,8957.74158847,6403.29279037,3070.06110398,6175.24851691,1461.27925187,
3336.50691137,6910.80070577,9967.11557409,3549.38634683,2606.03554605,1575.68922804,4906.98800552,679.32387231,
5310.51577039,6347.74789192,4980.14693502,7079.55612995,2304.94310865,253.435257548,6597.60226307,9160.92905888,
2656.51142121,1532.34098062,9800.24073027,4697.5946278,4549.56735803,895.369414414,81.2862597505,954.653076872,
9802.40869968,3068.94073212,4343.28386194,4288.43198471,9454.29190141,687.820715136,8555.10227894,1020.62527779,
8887.51891817,1622.30149751,5517.20514654,2660.16395818,9482.94902673,8842.14459278,3834.4145655,700.939818034,
8816.58299456,4351.4402913,3601.03531056,2832.71861204,5178.80878256,3393.39850047,3436.36898163,2336.76169531,
3094.84512625,9409.78159435,1337.62467857,5439.82313727,7485.84090711,3687.98225492,3607.41760755,4976.65341527,
7191.02440951,4980.12665621,3064.46805377
};


//this is an unused kernel
__global__ void kk(float * d)
{
	d[0]=kconst[0];}


__global__ void kernel(float*,float*,float*,int*,int*,int *,int,float *,int *);
float uniform(float start,float end)
{
	return start+(end-start)*rand()/RAND_MAX;
}

int main(int argc, char* argv[])
{
	int gridx = 179;
	int gridy = 1;
	int gridz = 1;
	int bx = 115;
	int by = 1;
	int bz = 1;

	dim3 gridDim(gridx,gridy,gridz);
	dim3 blockDim(bx,by,bz);

	//srand( (unsigned)time( NULL ) );
	srand(4);
	//declare variables
	int N=64,size=0,loop=0,globalN=1024*1024*128,globalspace;

	//initialize the variables
	N=gridx*gridy*gridz*bx*by*bz;
	loop=16;

	N=gridx*gridy*gridz*bx*by*bz*17;
	size=(N)*sizeof(float);
	globalspace=(globalN)*sizeof(float);
	//declare the float pointer variables
	float * f0,*fd0,*f1,*fd1,*f2,*fd2,*gf,*gf0;
	//parameter f0
	f0=(float*)malloc(size);
	memset(f0,0,size);
	hipMalloc((void**)&fd0,size);
	for(int i=0;i<N;i++)
	f0[i]=uniform(1,10000);
	hipMemcpy(fd0,f0,size,hipMemcpyHostToDevice);

	//parameter f1
	f1=(float*)malloc(size);
	memset(f1,0,size);
	hipMalloc((void**)&fd1,size);
	for(int i=0;i<N;i++)
	f1[i]=uniform(1,10000);
	hipMemcpy(fd1,f1,size,hipMemcpyHostToDevice);

	//parameter f2
	f2=(float*)malloc(size);
	memset(f2,0,size);
	hipMalloc((void**)&fd2,size);
	for(int i=0;i<N;i++)
	f2[i]=uniform(1,10000);
	hipMemcpy(fd2,f2,size,hipMemcpyHostToDevice);

	//global memory declare(float)
	gf=(float*)malloc(globalspace);
	memset(gf,0,globalspace);
	hipMalloc((void**)&gf0,globalspace);
	for(int i=0;i<globalN;i++)
		gf[i]=uniform(1,10000);
	hipMemcpy(gf0,gf,globalspace,hipMemcpyHostToDevice);

	//declare the int pointer variables
	int * d0,*dd0,*d1,*dd1,*d2,*dd2,*gd,*gd0;
	size=(N)*sizeof(int);

	globalspace=(globalN)*sizeof(int);
	//parameter d0
	d0=(int*)malloc(size);
	memset(d0,0,size);
	hipMalloc((void**)&dd0,size);
	for(int i=0;i<N;i++)
	d0[i]=uniform(1,10000);
	hipMemcpy(dd0,d0,size,hipMemcpyHostToDevice);

	//parameter d1
	d1=(int*)malloc(size);
	memset(d1,0,size);
	hipMalloc((void**)&dd1,size);
	for(int i=0;i<N;i++)
	d1[i]=uniform(1,10000);
	hipMemcpy(dd1,d1,size,hipMemcpyHostToDevice);

	//parameter d2
	d2=(int*)malloc(size);
	memset(d2,0,size);
	hipMalloc((void**)&dd2,size);
	for(int i=0;i<N;i++)
	d2[i]=uniform(1,10000);
	hipMemcpy(dd2,d2,size,hipMemcpyHostToDevice);

	//global memory declare(int)
	gd=(int*)malloc(globalspace);
	memset(gd,0,globalspace);
	hipMalloc((void**)&gd0,globalspace);
	for(int i=0;i<globalN;i++)
		gd[i]=uniform(1,10000);
	hipMemcpy(gd0,gd,globalspace,hipMemcpyHostToDevice);

	kernel<<<gridx*gridy,bx*by*bz>>>(fd0,fd1,fd2,dd0,dd1,dd2,loop,gf0,gd0);

	hipMemcpy(d2,dd2, size, hipMemcpyDeviceToHost);
	/*for(int i=0;i<N;i++)
		printf("%d	",d2[i]);
	printf("\n");*/
	hipMemcpy(f2,fd2, size, hipMemcpyDeviceToHost);
	/*for(int i=0;i<N;i++)
		printf("%f	",f2[i]);
	printf("\n");*/
}

__global__ void kernel(float* f0,float* f1,float* f2,int *d0,int *d1,int *d2,int loop,float* gf,int* gd)
{
	//declear the regs
	asm volatile(".reg .u32	%rt<4>;\n\
	.reg .u32 %rnt<4>;\n\
	.reg .u32 %rc<4>;\n\
	.reg .u32 %rnc<4>;\n\
	.reg .u32 %rg<14>;\n\
	.reg .u32 %rgtid;\n\
	.reg .u32 %rbtid;");
	//compute global threadID
	asm volatile("mov.u32	%rt1,%tid.x;\n\
	mov.u32	%rt2,%tid.y;\n\
	mov.u32	%rt3,%tid.z;\n\
	mov.u32	%rnt1,%ntid.x;\n\
	mov.u32	%rnt2,%ntid.y;\n\
	mov.u32	%rnt3,%ntid.z;\n\
	mov.u32	%rc1,%ctaid.x;\n\
	mov.u32	%rc2,%ctaid.y;\n\
	mov.u32	%rc3,%ctaid.z;\n\
	mov.u32	%rnc1,%nctaid.x;\n\
	mov.u32	%rnc2,%nctaid.y;\n\
	mov.u32	%rnc3,%nctaid.z;");	//compute threadID in a block
	asm volatile("mul.lo.u32	%rg1,%rnt1,%rnt2;\n\
	mul.lo.u32	%rg2,%rg1,%rt3;\n\
	mul.lo.u32	%rg3,%rt2,%rnt1;\n\
	add.u32	%rg4,%rt1,%rg3;\n\
	add.u32	%rg5,%rg4,%rg2;\n\
	mov.u32 %rbtid,%rg5;");
	//compute blockid in a grid
	asm volatile("mul.lo.u32	%rg6,%rnc1,%rnc2;\n\
	mul.lo.u32	%rg7,%rg6,%rc3;\n\
	mul.lo.u32	%rg8,%rc2,%rnc1;\n\
	add.u32	%rg9,%rc1,%rg8;\n\
	add.u32	%rg10,%rg9,%rg7;");
	//compute blocksize
	asm volatile("mul.lo.u32	%rg11,%rnt1,%rnt2;\n\
	mul.lo.u32	%rg12,%rg11,%rnt3;\n\
	mul.lo.u32	%rg13,%rg10,%rg12;\n\
	add.u32	%rgtid,%rg13,%rbtid;");
	asm volatile(".reg .u64	%rdf<3>;\n\
	.reg .u64	%rpf<3>;\n\
	.reg .u64	%rdd<3>;\n\
	.reg .u64	%rpd<3>;\n\
	.reg .pred	%p_<10>;\n\
	.reg .f32	%f_<3>;\n\
	.reg .s32	%d_<3>;\n\
	.reg .u64	%offset;\n\
	.reg .u64	%g_distance;\n\
	.reg .u32	%loop;\n\
	.reg .u32	%pass;\n\
	.reg .u64	%distance;\n\
	.reg .u64	%distance_mem;\n\
	.reg .u64	%gfp;\n\
	.reg .u64	%gdp;\n\
	.reg .u32	%counter;");
	asm volatile(".reg .u32	%rd<15>;\n\
	.reg .f32	%rf<16>;\n\
	.reg .f32	%rgf<32>;\n\
	.reg .u32	%rgd<32>;\n\
	.reg .b32	%rb<5>;\n\
	.reg .f64    %rdouble<8>;\n\
	.reg .f64	%rmadf<3>;\n\
	.reg .s32	%ropd3;\n\
	.reg .b32	%ropb<2>;\n\
	mov .b32	%ropb0,652;\n\
	mov .b32	%ropb1,256;\n\
	mov .u64	%distance,20585;\n\
	mov .u64	%distance_mem,262144;\n\
	mov .u64	%g_distance,65536;\n\
	mov .s32	%ropd3,5786;\n\
	mov .f64	%rmadf1,19.5923428225;\n\
	mov .f64     %rmadf2,23.9162632576;\n\
	mov .f64     %rmadf0,86.2923868728;");

	asm volatile(".reg .u64    %c_base;\n\
	.reg .u64    %c_offset;\n\
	.reg .u64    %c_mem;\n\
	mov.u64  %c_base,kconst;\n\
	mul.wide.u32   %c_offset,%rbtid,4;\n\
	add.u64  %c_mem,%c_base,%c_offset;");

	//get the loop number
	asm volatile("ld.param.u32   %loop, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_6];");//loop

	asm volatile("mul.wide.u32	%offset,%rgtid,4;");//get the address offset
//d0 data

	asm volatile("ld.param.u64	%rpd0, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_0];\n\
	add.u64	%rdd0,%rpd0,%offset;");
//d1 data

	asm volatile("ld.param.u64	%rpd1, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_1];\n\
	add.u64	%rdd1,%rpd1,%offset;");
//d2 data

	asm volatile("ld.param.u64	%rpd2, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_2];\n\
	add.u64	%rdd2,%rpd2,%offset;");
//f0 data

	asm volatile("ld.param.u64	%rpf0, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_3];\n\
	add.u64	%rdf0,%rpf0,%offset;");
//f1 data

	asm volatile("ld.param.u64	%rpf1, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_4];\n\
	add.u64	%rdf1,%rpf1,%offset;");
//f2 data

	asm volatile("ld.param.u64	%rpf2, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_5];\n\
	add.u64	%rdf2,%rpf2,%offset;");

	asm volatile("ld.param.u64	%gfp, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_7];\n\
	add.u64	%gfp,%gfp,%offset;");

	asm volatile("ld.param.u64	%gdp, [_Z6kernelPfS_S_PiS0_S0_iS_S0__param_8];\n\
	add.u64	%gdp,%gdp,%offset;");
	
//load the values of parameters
	asm volatile("ld.global.s32	%d_0,[%rdd0+0];");
	asm volatile("ld.global.s32	%d_1,[%rdd1+0];");
	asm volatile("ld.global.f32	%f_0,[%rdf0+0];");
	asm volatile("ld.global.f32	%f_1,[%rdf1+0];");


	asm volatile("mul.wide.u32	%distance,20585,4;\n\
	add.u64	%rdd0,%rdd0,%distance;\n\
	add.u64	%rdd1,%rdd1,%distance;\n\
	add.u64	%rdf0,%rdf0,%distance;\n\
	add.u64	%rdf1,%rdf1,%distance;");
	asm volatile("mov.u32	%counter,0;");
	asm volatile("$BB_LABEL0:");
	// instruction mix
	asm volatile("mov.u32 %pass,0;\n\
	fma.rn.f64	%rdouble0, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf0,%rdouble0;\n\
	lg2.approx.f32	%rf1, %f_1;\n\
	mad.lo.s32	%rd0, %d_0, %d_1, %ropd3;\n\
	mad.rn.f64	%rdouble1, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf2,%rdouble1;\n\
	lg2.approx.f32	%rf3, %f_0;\n\
	sqrt.rn.f32	%rf4, %f_1;\n\
	st.global.f32  [%gfp+0],%rgf19;\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	mul.f32	%rf5, %rf0, %rf1;\n\
	ld.const.f32	%rf0, [%c_mem+0];\n\
	cvt.f64.f32     %rmadf0, %rf0;\n\
	cvt.f64.f32     %rmadf1, %rf1;\n\
	cvt.f64.f32     %rmadf2, %rf2;\n\
	fma.rn.f64	%rdouble2, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf6,%rdouble2;\n\
	ld.const.f32	%rf3, [%c_mem+0];\n\
	xor.b32	%rb0, %ropb0, %ropb1;\n\
	ld.global.f32	%rgf11, [%gfp+0];\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	ld.global.f32	%rgf17, [%gfp+0];\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	mad.lo.s32	%rd1, %d_0, %d_1, %ropd3;\n\
	mad.lo.s32	%rd2, %d_0, %d_1, %ropd3;\n\
	ld.global.f32	%rgf8, [%gfp+0];\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	cvt.f64.f32     %rmadf0, %rf1;\n\
	cvt.f64.f32     %rmadf1, %rf2;\n\
	cvt.f64.f32     %rmadf2, %rf3;\n\
	mad.rn.f64	%rdouble3, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf7,%rdouble3;\n\
	mul.lo.s32	%rd3, %d_0, %d_1;\n\
	ld.const.f32	%rf1, [%c_mem+0];\n\
	neg.s32	%rd4, %d_0;\n\
	not.b32	%rb1, %ropb0;\n\
	cvt.f64.f32     %rmadf0, %rf2;\n\
	cvt.f64.f32     %rmadf1, %rf3;\n\
	cvt.f64.f32     %rmadf2, %rf4;\n\
	fma.rn.f64	%rdouble4, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf8,%rdouble4;\n\
	mad.lo.s32	%rd5, %rd0, %rd1, %rd2;\n\
	ld.global.f32	%rgf2, [%gfp+0];\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	cvt.f64.f32     %rmadf0, %rf3;\n\
	cvt.f64.f32     %rmadf1, %rf4;\n\
	cvt.f64.f32     %rmadf2, %rf5;\n\
	mad.rn.f64	%rdouble5, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf9,%rdouble5;\n\
	and.b32	%rb2, %ropb0, %ropb1;\n\
	cvt.f64.f32     %rmadf0, %rf4;\n\
	cvt.f64.f32     %rmadf1, %rf5;\n\
	cvt.f64.f32     %rmadf2, %rf6;\n\
	mad.rn.f64	%rdouble6, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf10,%rdouble6;\n\
	lg2.approx.f32	%rf11, %rf6;\n\
	cos.approx.f32	%rf12, %rf7;\n\
	rem.s32	%rd6, %rd1, %d_0;\n\
	rem.s32	%rd7, %rd2, %d_0;\n\
	mad.lo.s32	%rd8, %rd3, %rd4, %rd5;\n\
	cvt.f64.f32     %rmadf0, %rf7;\n\
	cvt.f64.f32     %rmadf1, %rf8;\n\
	cvt.f64.f32     %rmadf2, %rf9;\n\
	fma.rn.f64	%rdouble7, %rmadf0, %rmadf1, %rmadf2;\n\
	cvt.rn.f32.f64     %rf13,%rdouble7;\n\
	xor.b32	%rb3, %ropb0, %ropb1;\n\
	ex2.approx.f32	%rf14, %rf9;\n\
	ld.global.f32	%rgf31, [%gfp+0];\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	ld.global.f32	%rgf25, [%gfp+0];\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	shl.b32	%rb4, %ropb0, %ropb1;\n\
	max.s32	%rd9, %rd4, %rd5;\n\
	sad.s32	%rd10, %rd5, %rd6, %rd7;\n\
	ld.global.f32	%rgf20, [%gfp+0];\n\
	add.u64 %gfp,%gfp,%g_distance;\n\
	div.s32	%rd11, %rd6, %d_0;\n\
	mad.lo.s32	%rd12, %rd7, %rd8, %rd9;\n\
	min.s32	%rd13, %rd8, %rd9;\n\
	");
	asm volatile("add.u32	%counter,%counter,1;\n\
	setp.ge.u32	 %p_0, %counter, %loop;\n\
	@!%p_0	bra	$BB_LABEL0;");

	asm volatile("st.global.f32 [%rdf2+0],%rf15;\n\
	st.global.s32 [%rdd2+0],%rd14;");
	asm volatile("add.u64	%rdf2,%rdf2,%distance;\n\
	add.u64	%rdd2,%rdd2,%distance;");

	//end basic block;
	asm volatile("$BB_LABEL1:");
	asm volatile("exit;");
}
